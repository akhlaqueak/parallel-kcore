#include "hip/hip_runtime.h"

#include "../inc/device_funcs.h"
#include "stdio.h"
#include "buffer.cc"

#define SHARED_MEMORY_BANKS 32
#define LOG_MEM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_MEM_BANKS)

__device__ void scanBlock(unsigned int *input, unsigned int *output)
{
	__shared__ unsigned int temp[BLK_DIM*2];// allocated on invocation

	unsigned int ai = THID;
	unsigned int bi = THID + (BLK_DIM / 2);
	unsigned int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	unsigned int bankOffsetB = CONFLICT_FREE_OFFSET(bi);


    temp[ai + bankOffsetA] = input[ai];
    temp[bi + bankOffsetB] = input[bi];


	unsigned int offset = 1;
	for (unsigned int d = BLK_DIM >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (THID < d)
		{
			unsigned int ai = offset * (2 * THID + 1) - 1;
			unsigned int bi = offset * (2 * THID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (THID == BLK_DIM-1) {
		temp[THID-1 + CONFLICT_FREE_OFFSET(THID - 1)] = 0; // clear the last element
	}

	for (unsigned int d = 1; d < BLK_DIM; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (THID < d)
		{
			unsigned int ai = offset * (2 * THID + 1) - 1;
			unsigned int bi = offset * (2 * THID + 2) - 1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			unsigned int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

    output[ai] = temp[ai + bankOffsetA];
    output[bi] = temp[bi + bankOffsetB];

}

__device__ void selectNodesAtLevel(unsigned int *degrees, unsigned int V, unsigned int* shBuffer, unsigned int* glBuffer, unsigned int* bufTailPtr, unsigned int level){

    unsigned int glThreadIdx = blockIdx.x * BLK_DIM + THID; 

    __shared__ bool predicate[BLK_DIM];
    __shared__ unsigned int addresses[BLK_DIM];
    __shared__ unsigned int scannedAddresses[BLK_DIM];
    __shared__ unsigned int bTail;
    
    for(unsigned int base = 0; base < V; base += N_THREADS){
        
        unsigned int v = base + glThreadIdx; 

        // all threads should get some value, if vertices are less than n_threads, rest of the threads get zero
        predicate[THID] = (v<V)? (degrees[v] == level) : 0;

        addresses[THID] = predicate[THID];

        scanBlock(addresses, scannedAddresses);

        
        if(THID == BLK_DIM - 1){  
            int nv =  scannedAddresses[THID] + predicate[THID];            
            bTail = nv>0? atomicAdd(bufTailPtr, nv) : 0;
            
        }

        // this sync is necessary so that memory is allocated before writing to buffer
        __syncthreads();
        
        scannedAddresses[THID] += bTail;
        
        if(predicate[THID])
            writeToBuffer(shBuffer, glBuffer, scannedAddresses[THID], v);
        
        __syncthreads();
            
    }
}




__device__ void syncBlocks(unsigned int* blockCounter){
    
    if (THID==0)
    {
        atomicAdd(blockCounter, 1);
        __threadfence();
        while(ldg(blockCounter)<BLK_NUMS){
            // number of blocks can't be greater than SMs, else it'll cause infinite loop... 
            // printf("%d ", blockCounter[0]);
        };// busy wait until all blocks increment
    }   
    __syncthreads();
}

__global__ void PKC(G_pointers d_p, unsigned int *global_count, int level, int V, 
                    unsigned int* blockCounter, unsigned int* glBuffers){


    __shared__ unsigned int shBuffer[MAX_NV];
    __shared__ unsigned int bufTail;
    __shared__ unsigned int base;
    __shared__ unsigned int lock;
    unsigned int warp_id = THID / 32;
    unsigned int lane_id = THID % 32;
    unsigned int i;

    bufTail = 0;
    base = 0;
    lock = 0;
    unsigned int* glBuffer = glBuffers + blockIdx.x * GLBUFFER_SIZE; 

    __syncthreads();

    selectNodesAtLevel(d_p.degrees, V, shBuffer, glBuffer, &bufTail, level);

    syncBlocks(blockCounter);

    // bufTail is being incrmented within the loop, 
    // warps should process all the nodes added during the execution of loop
    // for that purpose e_processes is introduced, is incremented whenever a warp takes a job. 
    
    
    // for(unsigned int i = warp_id; i<bufTail ; i = warp_id + base){
    // this for loop is a wrong choice, as many threads will exit from the loop checking the condition
    while(true){
        __syncthreads(); //syncthreads must be executed by all the threads, so can't put after break or continue...

        if(base == bufTail) break;

        i = base + warp_id;
        
        if(THID == 0){
            base += WARPS_EACH_BLK;
            if(bufTail < base )
                base = bufTail;
        }
        __syncthreads();
        if(i >= bufTail) continue; // this warp won't have to do anything     
        
        
        unsigned int v, start, end;

        v = readFromBuffer(shBuffer, glBuffer, i);
        start = d_p.neighbors_offset[v];
        end = d_p.neighbors_offset[v+1];


        while(true){
            __syncwarp();

            if(start >= end) break;

            unsigned int j = start + lane_id;
            start += WARP_SIZE;
            if(j >= end) continue;

            unsigned int u = d_p.neighbors[j];
            if(ldg(d_p.degrees+u) > level){
                unsigned int a = atomicSub(d_p.degrees+u, 1);
            
                if(a == level+1){
                    unsigned int loc = atomicAdd(&bufTail, 1);
                    writeToBuffer(shBuffer, glBuffer, loc, u);
                }

                if(a <= level){
                    // node degree became less than the level after decrementing... 
                    atomicAdd(d_p.degrees+u, 1);
                }
            }
        }

    }


    if(THID == 0 ){
        if(bufTail>0) atomicAdd(global_count, bufTail); // atomic since contention among blocks
        // if(glBuffer!=NULL) free(glBuffer);
    }

}


