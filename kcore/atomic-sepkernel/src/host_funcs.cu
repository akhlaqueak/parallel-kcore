
#include "../inc/host_funcs.h"
#include "../inc/gpu_memory_allocation.h"

inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout<<hipGetErrorString(code)<<std::endl;
        exit(-1);
    }
}
void copy_graph_to_gpu(Graph data_graph, G_pointers &data_pointers){
    malloc_graph_gpu_memory(data_graph,data_pointers);
}
void find_kcore(string data_file,bool write_to_disk){
    cout<<"start loading graph file from disk to memory..."<<endl;
    
    Graph data_graph(data_file);

    cout<<"graph loading complete..."<<endl;
    G_pointers data_pointers;


    cout<<"start copying graph to gpu..."<<endl;
    malloc_graph_gpu_memory(data_graph, data_pointers);
    cout<<"end copying graph to gpu..."<<endl;

    unsigned int level = 0;
    unsigned int count = 0;
    unsigned int* global_count  = NULL;
    unsigned int* bufTails  = NULL;
    unsigned int* glBuffers     = NULL;

    chkerr(hipMalloc(&global_count, sizeof(unsigned int)));
    chkerr(hipMalloc(&bufTails, sizeof(unsigned int)*BLK_NUMS));
    hipMemset(global_count, 0, sizeof(unsigned int));
    
    
    
    size_t limit = 0;
    hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    
    cout<<"default limit is: "<<limit<<endl;
    
    limit = 1024*1024*1024ULL;
    chkerr(hipDeviceSetLimit(hipLimitMallocHeapSize, limit));
    limit = 0;
    hipDeviceGetLimit(&limit, hipLimitMallocHeapSize);
    chkerr(hipMalloc(&glBuffers,sizeof(unsigned int)*BLK_NUMS*GLBUFFER_SIZE));
    
    cout<<"new limit is: "<<limit<<endl;
    
    
	cout<<"Entering in while"<<endl;

    auto start = chrono::steady_clock::now();
    while(count < data_graph.V){
        hipMemset(bufTails, 0, sizeof(unsigned int)*BLK_NUMS);
        // chkerr(hipDeviceSynchronize());
        selectNodesAtLevel<<<BLK_NUMS, BLK_DIM>>>(data_pointers.degrees, level, data_graph.V, bufTails, glBuffers);

        processNodes<<<BLK_NUMS, BLK_DIM>>>(data_pointers, level, data_graph.V, bufTails, glBuffers, global_count);
        // test<<<BLK_NUMS, BLK_DIM>>>(data_pointers.degrees);
        chkerr(hipMemcpy(&count, global_count, sizeof(unsigned int), hipMemcpyDeviceToHost));    
        
        cout<<"*********Completed level: "<<level<<", global_count: "<<count<<" *********"<<endl;
        level++;
    }
    auto end = chrono::steady_clock::now();
    
    
    cout << "Elapsed Time: "
    << chrono::duration_cast<chrono::milliseconds>(end - start).count() << endl;
    cout <<"MaxK: "<<level-1<<endl;
    
    
	get_results_from_gpu(data_graph, data_pointers);
    
    hipFree(glBuffers);
    free_graph_gpu_memory(data_pointers);
    if(write_to_disk){
        cout<<"Writing kcore to disk started... "<<endl;
        write_kcore_to_disk(data_graph.degrees, data_graph.V, data_file);
        cout<<"Writing kcore to disk completed... "<<endl;
    }

}
