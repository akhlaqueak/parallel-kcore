#include "hip/hip_runtime.h"

#include "../inc/device_funcs.h"
#include "../inc/scans.h"
#include "stdio.h"


// __device__ inline unsigned int getWriteLoc(unsigned int* bufTail){
//     return atomicAdd(bufTail, 1);
// }

__device__ void writeToBuffer(unsigned int* shBuffer,    unsigned int* glBuffer, unsigned int loc, unsigned int v){
    assert(loc < GLBUFFER_SIZE + MAX_NV);
    if(loc < MAX_NV)
        shBuffer[loc] = v;
    else
        glBuffer[loc-MAX_NV] = v;
}


__device__ unsigned int readFromBuffer(unsigned int* shBuffer,   unsigned int* glBuffer, unsigned int loc){
    assert(loc < GLBUFFER_SIZE + MAX_NV);
    return ( loc < MAX_NV ) ? shBuffer[loc] : glBuffer[loc-MAX_NV]; 
}



__device__ inline bool allocationRequired( unsigned int* glBuffer, unsigned int loc, unsigned int dim){
    return (THID%dim == dim-1 && // last thread of warp or block
        glBuffer == NULL && // global buffer is not allocated before
        loc >= MAX_NV
    );
}
__device__ inline void allocateMemory( unsigned int** glBufferPtr){
        glBufferPtr[0] = ( unsigned int*) malloc(sizeof(unsigned int) * GLBUFFER_SIZE);
        // printf("allocated %d %d\n", blockIdx.x, THID);
        assert(glBufferPtr[0]!=NULL);        
}

__device__ void allocateMemoryMutex( unsigned int** glBufferPtr, unsigned int loc, volatile unsigned int* lock){
    if(atomicExch((unsigned int*)lock, 1) == 0){        
        // printf("mutex %d %d\n", blockIdx.x, THID);
        allocateMemory(glBufferPtr);
        lock[0] = 2; // not necessary to do it atomically, since it's the only thread in critical section
        __threadfence_block(); // it ensures the writes done by this thread are visible by all other threads in the block
    }
    while(lock[0]!=2);
}    

__device__ void syncBlocks(volatile unsigned int* blockCounter){
    
    if (THID==0)
    {
        atomicAdd((unsigned int*)blockCounter, 1);
        __threadfence();
        while(blockCounter[0]<BLK_NUMS){
            // number of blocks can't be greater than SMs, else it'll cause infinite loop... 
            // printf("%d ", blockCounter[0]);
        };// busy wait until all blocks increment
    }
    

    
    __syncthreads();
}

__global__ void PKC(G_pointers d_p, unsigned int *global_count, int level, int V, volatile unsigned int* blockCounter){
    
    
    __shared__ unsigned int shBuffer[MAX_NV];
    __shared__ unsigned int bufTail;
    __shared__  unsigned int* glBuffer;
    __shared__ unsigned int base;
    __shared__ unsigned int predicate[BLK_DIM];
    __shared__ unsigned int temp[BLK_DIM];
    __shared__ unsigned int addresses[BLK_DIM];
    __shared__ volatile unsigned int allocLock;
    __shared__ volatile unsigned int readLock;

    unsigned int warp_id = THID / 32;
    unsigned int lane_id = THID % 32;
    unsigned int i;

    bufTail = 0;
    glBuffer = NULL;
    base = 0;
    predicate[THID] = 0;
    allocLock = 0;
    readLock = 0;

    compactBlock(d_p.degrees, V, shBuffer, &glBuffer, &bufTail, level);
    // if(level == 1 && THID == 0) printf("%d ", bufTail);

    __syncthreads();

    
    // bufTail is being incremented within the loop, 
    // warps should process all the nodes added during the execution of loop
    // for that purpose base is introduced, is incremented whenever a warp takes a job.
    
    // todo: busy waiting on several blocks

    syncBlocks(blockCounter);
    // bufTail = 10;
    // for(unsigned int i = warp_id; i<bufTail ; i += WARPS_EACH_BLK){
    // this for loop is a wrong choice, as many threads might exit from the loop checking the condition     
    while(true){
        __syncthreads(); //syncthreads must be executed by all the threads, so can't put after break or continue...
        if(base == bufTail) break;
        i = base + warp_id;
        
        if(THID == 0){
            base += WARPS_EACH_BLK;
            if(bufTail < base )
                base = bufTail;
        }
        __syncthreads();
        if(i >= bufTail) continue; // this warp won't have to do anything 

        
        unsigned int v = readFromBuffer(shBuffer, glBuffer, i);
        unsigned int start = d_p.neighbors_offset[v];
        unsigned int end = d_p.neighbors_offset[v+1];
        unsigned int b1 = start;
        // for(int j = start + lane_id; j<end ; j+=32){
        // the for loop may leave some of the threads inactive in its last iteration
        // following while loop will keep all threads active until the continue condition
        while(true){
            __syncwarp();

            compactWarp(temp+(warp_id*WARP_SIZE), addresses+(warp_id*WARP_SIZE), predicate+(warp_id*WARP_SIZE), shBuffer, &glBuffer, &bufTail, &allocLock);
            
            if(b1 >= end) break;

            unsigned int j = b1 + lane_id;
            b1 += WARP_SIZE;
            if(j >= end) continue;

            unsigned int u = d_p.neighbors[j];
            if(d_p.degrees[u] > level){
                unsigned int a = atomicSub(d_p.degrees+u, 1);
            
                if(a == level+1){
                    temp[THID] = u;
                    predicate[THID] = 1;
                    // unsigned int loc = getWriteLoc(&bufTail);
                    // writeToBuffer(shBuffer, &glBuffer, loc, u);
                }

                if(a <= level){
                    // node degree became less than the level after decrementing... 
                    atomicAdd(d_p.degrees+u, 1);
                }
                __threadfence();
            }
        }        
    }
    
    __syncthreads();

    if(THID == 0 && bufTail!=0){
        atomicAdd(global_count, bufTail); // atomic since contention among blocks
        if(glBuffer!=NULL) free((unsigned int*)glBuffer);
    }

}


