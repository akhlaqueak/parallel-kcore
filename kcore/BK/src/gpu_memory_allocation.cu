
#include "../inc/gpu_memory_allocation.h"

void malloc_graph_gpu_memory(Graph &g,G_pointers &p){
    chkerr(hipMalloc(&(p.neighbors),g.neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors,g.neighbors,g.neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset,g.neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degrees),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.degrees,g.degrees,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degOrder),(g.V)*sizeof(unsigned int)));
    chkerr(hipMalloc(&(p.total),sizeof(unsigned int)));
    hipMemset(p.total, 0, sizeof(unsigned int));
    p.V = g.V;
    // std::cout<<"memory graph p = "<<p.neighbors[0]<<endl;
}
void recodedGraphCopy(Graph &g, G_pointers &p, Subgraphs** sg1, Subgraphs** sg2){
    chkerr(hipMalloc(&(p.neighbors),g.neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors,g.neighbors,g.neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset,g.neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degrees),(g.V)*sizeof(unsigned int)));
    chkerr(hipMalloc(&(p.total),sizeof(unsigned int)));
    chkerr(hipMemset(p.total, 0, sizeof(unsigned int)));
    chkerr(hipMemcpy(p.degrees,g.degrees,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));
    p.V = g.V;
    chkerr(hipMallocManaged(sg1, BLK_NUMS*sizeof(Subgraphs)));
    for(int i=0;i<BLK_NUMS; i++){
        chkerr(hipMalloc(&(sg1[0][i].offsets), NSUBS*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg1[0][i].vertices), NSUBS*1000*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg1[0][i].labels), NSUBS*1000*sizeof(char)));
    }
    chkerr(hipMallocManaged(sg2, BLK_NUMS*sizeof(Subgraphs)));
    for(int i=0;i<BLK_NUMS; i++){
        chkerr(hipMalloc(&(sg2[0][i].offsets), NSUBS*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg2[0][i].vertices), NSUBS*1000*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg2[0][i].labels), NSUBS*1000*sizeof(char)));
    }
    for(int i=0;i<BLK_NUMS; i++){
        chkerr(hipMalloc(&(sg1[0][i].tempv), TEMPSIZE*WARPS_EACH_BLK*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg1[0][i].templ), TEMPSIZE*WARPS_EACH_BLK*sizeof(char)));
    }
}
void get_results_from_gpu(Graph &g,G_pointers &p){
    chkerr(hipMemcpy(g.degrees,p.degrees,(g.V)*sizeof(unsigned int),hipMemcpyDeviceToHost));    
}

void free_graph_gpu_memory(G_pointers &p){
    chkerr(hipFree(p.neighbors));
    chkerr(hipFree(p.neighbors_offset));
    chkerr(hipFree(p.degrees));
}
