
#include "../inc/gpu_memory_allocation.h"

void malloc_graph_gpu_memory(Graph &g,G_pointers &p){
    chkerr(hipMalloc(&(p.neighbors),g.neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors,g.neighbors,g.neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset,g.neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degrees),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.degrees,g.degrees,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degOrder),(g.V)*sizeof(unsigned int)));
    p.V = g.V;
    // std::cout<<"memory graph p = "<<p.neighbors[0]<<endl;
}
void recodedGraphCopy(Graph &g, G_pointers &p, Subgraphs** sg){
    chkerr(hipMalloc(&(p.neighbors),g.neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors,g.neighbors,g.neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset,g.neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degrees),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.degrees,g.degrees,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));
    p.V = g.V;
    chkerr(hipMalloc(sg, BLK_NUMS*sizeof(Subgraphs)));

    for(int i=0;i<BLK_NUMS; i++){
        chkerr(hipMalloc(&(sg[i].offsets), NSUBS*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg[i].vertices), NSUBS*1000*sizeof(unsigned int)));
        chkerr(hipMalloc(&(sg[i].labels), NSUBS*1000*sizeof(char)));
    }
}
void get_results_from_gpu(Graph &g,G_pointers &p){
    chkerr(hipMemcpy(g.degrees,p.degrees,(g.V)*sizeof(unsigned int),hipMemcpyDeviceToHost));    
}

void free_graph_gpu_memory(G_pointers &p){
    chkerr(hipFree(p.neighbors));
    chkerr(hipFree(p.neighbors_offset));
    chkerr(hipFree(p.degrees));
}
