#include "hip/hip_runtime.h"

#include "../inc/device_funcs.h"
#include "stdio.h"
#include "buffer.cc"
__global__ void BK(G_pointers dp, Subgraphs* subgs, unsigned int base){
    __shared__ Subgraphs sg;
    __shared__ unsigned int vtail, otail;
    // vtail: vertices tail, a subgraph vertices stored based on an atomic increment to it
    //          labels also use the same vtail
    // otail: offset tail, two consective values represent start and end of a subgraph.
    //          it's always atomically incremented by 2.

    unsigned int warpid = WARPID;
    unsigned int laneid = LANEID;
    if(THID==0){
        sg = subgs[BLKID];
        base += BLKID*SUBG;
        vtail = 0;
        otail = 0;
    }
    __syncthreads();

    // create subgraphs... 
    unsigned int u;
    unsigned int v = base+warpid;
    unsigned int start = dp.neighbors_offset[v];
    unsigned int end = dp.neighbors_offset[v+1];
    unsigned int len = end-start+1; // number of neighbors + v itself
    unsigned int loc;
    if(laneid == 0){
        loc = atomicAdd(&vtail, len);
        sg.vertices[loc] = v;
        sg.labels[loc++] = R;
        
        unsigned int st = atomicAdd(&otail, 2);
        sg.offsets[st] = loc;
        sg.offsets[st+1] = loc+len; 
        printf("%d-%d", st, loc);
    }
    loc = __shfl_sync(FULL, loc, 0);
    for(;start<end; start+=32, loc+=32){
        u = dp.neighbors[start+laneid];
        sg.vertices[loc+laneid] = u;
        if(u < v){sg.labels[loc+laneid] = X;}
        else {sg.labels[loc+laneid] = P;}
    }
    for(int i=0;i<otail;i+=2){
        unsigned int st = sg.offsets[i];
        unsigned int en = sg.offsets[i+1];
    }
}



__global__ void selectNodesAtLevel(unsigned int *degrees, unsigned int level, unsigned int V, 
                 unsigned int* bufTails, unsigned int* glBuffers){

    __shared__ unsigned int* glBuffer; 
    __shared__ unsigned int bufTail; 
    
    if(THID == 0){
        bufTail = 0;
        glBuffer = glBuffers + blockIdx.x*GLBUFFER_SIZE;
    }
    __syncthreads();

    unsigned int global_threadIdx = blockIdx.x * blockDim.x + threadIdx.x; 
    for(unsigned int base = 0; base < V; base += N_THREADS){
        
        unsigned int v = base + global_threadIdx; 

        if(v >= V) continue;

        if(degrees[v] == level){
            unsigned int loc = atomicAdd(&bufTail, 1);
            writeToBuffer(glBuffer, loc, v);
        }
    }

    __syncthreads();

    if(THID == 0) 
    {
        bufTails [blockIdx.x] = bufTail;
    }
}




__global__ void processNodes(G_pointers d_p, int level, int V, 
                    unsigned int* bufTails, unsigned int* glBuffers, 
                    unsigned int *global_count){

    __shared__ unsigned int bufTail;
    __shared__ unsigned int* glBuffer;
    __shared__ unsigned int base;
    unsigned int warp_id = THID / 32;
    unsigned int lane_id = THID % 32;
    unsigned int regTail;
    unsigned int i;
    if(THID==0){
        bufTail = bufTails[blockIdx.x];
        base = 0;
        glBuffer = glBuffers + blockIdx.x*GLBUFFER_SIZE; 
        assert(glBuffer!=NULL);
    }

    // bufTail is being incrmented within the loop, 
    // warps should process all the nodes added during the execution of loop
    
    // for(unsigned int i = warp_id; i<bufTail ; i +=warps_each_block ){
    // this for loop is a wrong choice, as many threads will exit from the loop checking the condition
    while(true){
        __syncthreads(); //syncthreads must be executed by all the threads
        if(base == bufTail) break; // all the threads will evaluate to true at same iteration
        i = base + warp_id;
        regTail = bufTail;
        __syncthreads();

        if(i >= regTail) continue; // this warp won't have to do anything            

        if(THID == 0){
            // base += min(WARPS_EACH_BLK, regTail-base)
            // update base for next iteration
            base += WARPS_EACH_BLK;
            if(regTail < base )
                base = regTail;
        }
        //bufTail is incremented in the code below:

        unsigned int v = readFromBuffer(glBuffer, i);
        unsigned int start = d_p.neighbors_offset[v];
        unsigned int end = d_p.neighbors_offset[v+1];


        while(true){
            __syncwarp();

            if(start >= end) break;

            unsigned int j = start + lane_id;
            start += WARP_SIZE;
            if(j >= end) continue;

            unsigned int u = d_p.neighbors[j];
            if(*(d_p.degrees+u) > level){
                
                unsigned int a = atomicSub(d_p.degrees+u, 1);
            
                if(a == level+1){
                    unsigned int loc = atomicAdd(&bufTail, 1);

                    writeToBuffer(glBuffer, loc, u);
                }

                if(a <= level){
                    // node degree became less than the level after decrementing... 
                    atomicAdd(d_p.degrees+u, 1);
                }
            }
        }

    }

    if(bufTail>0){
        if(THID == 0)
            base = atomicAdd(global_count, bufTail); // atomic since contention among blocks
        __syncthreads();
        // Store degeneracy order... 
        for(int i=THID; i<bufTail; i+=BLK_DIM){
            // d_p.degOrder[i] = glBuffer[i-base]; // nedds to process it again if done this way
            d_p.degOrder[ glBuffer[i] ] = atomicAdd(&base, 1);
        }
    }
}
