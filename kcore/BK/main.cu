#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>


#include "../inc/gpu_memory_allocation.h"
#include "../inc/device_funcs.h"

int find_kcore(Graph &g,bool write_to_disk){

    G_pointers dp;


    cout<<"Device Copy Started"<<endl;
    malloc_graph_gpu_memory(g, dp);
    cout<<"Device Copy Done"<<endl;
    unsigned int V = g.V;
    unsigned int level = 0;
    unsigned int count = 0;
    unsigned int* global_count  = NULL;
    unsigned int* bufTails  = NULL;
    unsigned int* glBuffers     = NULL;

    chkerr(hipMalloc(&global_count, sizeof(unsigned int)));
    chkerr(hipMalloc(&bufTails, sizeof(unsigned int)*BLK_NUMS));
    hipMemset(global_count, 0, sizeof(unsigned int));
    chkerr(hipMalloc(&glBuffers,sizeof(unsigned int)*BLK_NUMS*GLBUFFER_SIZE));
    
    
    
	cout<<"K-core Computation Started"<<endl;

    auto tick = chrono::steady_clock::now();
    while(count < g.V){
        hipMemset(bufTails, 0, sizeof(unsigned int)*BLK_NUMS);

        selectNodesAtLevel<<<BLK_NUMS, BLK_DIM>>>(dp.degrees, level, 
                        g.V, bufTails, glBuffers);

        processNodes<<<BLK_NUMS, BLK_DIM>>>(dp, level, g.V, 
                        bufTails, glBuffers, global_count);

        chkerr(hipMemcpy(&count, global_count, sizeof(unsigned int), hipMemcpyDeviceToHost));    
        cout<<"*********Completed level: "<<level<<", global_count: "<<count<<" *********"<<endl;
        level++;
    }
	cout<<"K-core Computation Done"<<endl;
    cout<<"KMax: "<< level-1 <<endl;
    Graph gRec(g); // copy constructor overloaded... it allocates array for degree, neighbors... 
  
    
    unsigned int rec[g.V];
    chkerr(hipMemcpy(&rec, dp.degOrder, sizeof(unsigned int)*V, hipMemcpyDeviceToHost));    
    
    for(int i=0;i<g.V;i++)
        gRec.degrees[rec[i]] = g.degrees[i];

    gRec.neighbors_offset[0] = 0;
    std::partial_sum(gRec.degrees, gRec.degrees+V, gRec.neighbors_offset+1);

    for(int v=0;v<V;v++){
        unsigned int recv = rec[v];
        unsigned int start = gRec.neighbors_offset[recv];
        unsigned int end = gRec.neighbors_offset[recv+1];
        for (int j=g.neighbors_offset[v], k=start; j<g.neighbors_offset[v+1]; j++, k++){
            gRec.neighbors[k] = rec[g.neighbors[j]];
        }
        std::sort(gRec.neighbors+start, gRec.neighbors+end);

    }
    cout<<"Reordering Time: "<<chrono::duration_cast<chrono::milliseconds>(chrono::steady_clock::now()-tick).count()<<endl;

    
    // cout << "Elapsed Time: "
    // << chrono::duration_cast<chrono::milliseconds>(end - start).count() << endl;
    // cout <<"MaxK: "<<level-1<<endl;
    
    
	// get_results_from_gpu(g, dp);
    
    hipFree(glBuffers);
    free_graph_gpu_memory(dp);
    // if(write_to_disk){
    //     cout<<"Writing kcore to disk started... "<<endl;
    //     g.writeKCoreToDisk(data_file);
    //     cout<<"Writing kcore to disk completed... "<<endl;
    // }

    return chrono::duration_cast<chrono::milliseconds>(chrono::steady_clock::now() - tick).count();

}


int main(int argc, char *argv[]){
    if (argc < 2) {
        cout<<"Please provide data file"<<endl;
        exit(-1);
    }
    std::string data_file = argv[1];
    bool write_to_disk = false;

    cout<<"Loading Started"<<endl;    
    Graph g(data_file);
    cout<<"Loading Done"<<endl;
    
    vector<int> et;
    for(int i=0;i<REP; i++){
        cout<<"Running iteration: "<<i+1<<endl;
        int t = find_kcore(g, write_to_disk);
        et.push_back(t);
    }
    cout << data_file << " Elapsed Time: ";

    for(auto t: et)
        cout<<t<<" ";
    cout<<(double)accumulate(et.begin(), et.end(), 0)/et.size();
    cout<<endl;
    return 0;
}
