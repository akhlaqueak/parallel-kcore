#include "hip/hip_runtime.h"

#include "../inc/device_funcs.h"
#include "stdio.h"
#include "buffer.cc"
#include "scans.cc"


__global__ void selectNodesAtLevel(unsigned int* degrees, unsigned int *bufTails, int level, int V, 
    unsigned int* glBuffers){

    __shared__ bool predicate[BLK_DIM];
    __shared__ unsigned int temp[BLK_DIM];
    __shared__ unsigned int addresses[BLK_DIM];
    __shared__ unsigned int bufTail;
    __shared__ unsigned int* glBuffer;
    __shared__ unsigned int* shBuffer;
    if(THID==0){
        bufTail = 0;
        glBuffer = glBuffers+(blockIdx.x*GLBUFFER_SIZE);
    }

    unsigned int glThreadIdx = blockIdx.x * BLK_DIM + THID; 

    for(unsigned int base = 0; base < V; base += N_THREADS){
        
        unsigned int v = base + glThreadIdx; 

        // all threads should get some value, if vertices are less than n_threads, rest of the threads get zero
        predicate[THID] = (v<V)? (degrees[v] == level) : 0;
        if(predicate[THID]) temp[THID] = v;

        compactWarp(predicate, addresses, temp, shBuffer, glBuffer, &bufTail);        
        
        __syncthreads();
            
    }
    if(THID==0){
        bufTails[blockIdx.x] = bufTail;
    }
}






__global__ void PKC(G_pointers d_p, unsigned int *global_count, int level, int V, 
                    unsigned int* bufTails, unsigned int* glBuffers){
    
    __shared__ volatile unsigned int addresses[BLK_DIM];
    __shared__ bool predicate[BLK_DIM];
    __shared__ unsigned int temp[BLK_DIM];
    __shared__ unsigned int *shBuffer;
    __shared__ unsigned int bufTail;
    __shared__ volatile unsigned int base;
    __shared__ unsigned int* glBuffer;
    unsigned int warp_id = THID / 32;
    unsigned int lane_id = THID % 32;
    unsigned int i;
    
    if(THID==0){
        bufTail = bufTails[blockIdx.x];
        base = 0;
        glBuffer = glBuffers + blockIdx.x * GLBUFFER_SIZE; 
    }

    __syncthreads();
    

    
    predicate[THID] = 0;
    // bufTail is being incrmented within the loop, 
    // warps should process all the nodes added during the execution of loop
    // for that purpose e_processes is introduced, is incremented whenever a warp takes a job. 
    
    
    // for(unsigned int i = warp_id; i<bufTail ; i = warp_id + base){
    // this for loop is a wrong choice, as many threads will exit from the loop checking the condition
    while(true){
        __syncthreads(); //syncthreads must be executed by all the threads, so can't put after break or continue...

        if(base == bufTail) break;

        i = base + warp_id;
        
        if(THID == 0){
            base += WARPS_EACH_BLK;
            if(bufTail < base )
                base = bufTail;
        }
        // __syncthreads();
        if(i >= bufTail) continue; // this warp won't have to do anything     
        
        
        unsigned int v, start, end;

        v = readFromBuffer(shBuffer, glBuffer, i);
        start = d_p.neighbors_offset[v];
        end = d_p.neighbors_offset[v+1];


        while(true){
            // __syncwarp();

            compactWarp(predicate, addresses, temp, shBuffer, glBuffer, &bufTail);
            if(start >= end) break;

            unsigned int j = start + lane_id;
            start += WARP_SIZE;
            if(j >= end) continue;

            unsigned int u = d_p.neighbors[j];
            if(ldg(d_p.degrees+u) > level){
                unsigned int a = atomicSub(d_p.degrees+u, 1);
            
                if(a == level+1){
                    temp[THID] = u;
                    predicate[THID] = 1;
                    // unsigned int loc = atomicAdd(&bufTail, 1);
                    // writeToBuffer(shBuffer, glBuffer, loc, u);
                }

                if(a <= level){
                    // node degree became less than the level after decrementing... 
                    atomicAdd(d_p.degrees+u, 1);
                }
            }
        }

    }
    __syncthreads();

    if(THID == 0 ){
        if(bufTail>0) atomicAdd(global_count, bufTail); // atomic since contention among blocks
        // if(glBuffer!=NULL) free(glBuffer);
    }

}


