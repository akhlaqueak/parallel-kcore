#include "hip/hip_runtime.h"

#include "../inc/device_funcs.h"
#include "stdio.h"
#include "buffer.cc"


__device__ void selectNodesAtLevel(unsigned int *degrees, unsigned int V, unsigned int* shBuffer, unsigned int* glBuffer, unsigned int* bufTail, unsigned int level){
    unsigned int global_threadIdx = blockIdx.x * blockDim.x + threadIdx.x; 
    for(unsigned int base = 0; base < V; base += N_THREADS){
        
        unsigned int v = base + global_threadIdx; 

        if(v >= V) continue;

        if(degrees[v] == level){
            unsigned int loc = atomicAdd(bufTail, 1);
            writeToBuffer(shBuffer, glBuffer, loc, v);
        }
    }
}



__device__ void syncBlocks(unsigned int* blockCounter){
    
    if (THID==0)
    {
        atomicAdd(blockCounter, 1);
        __threadfence();
        
        while(ldg(blockCounter) < BLK_NUMS){
            // number of blocks can't be greater than SMs, else it'll cause infinite loop... 
            // printf("%d ", blockCounter[0]);
        };// busy wait until all blocks increment
    }   
    __syncthreads();
}

__global__ void PKC(G_pointers d_p, unsigned int *global_count, int level, int V, 
                    unsigned int* blockCounter, unsigned int* glBuffers){


    __shared__ unsigned int shBuffer[MAX_NV];
    __shared__ unsigned int bufTail;
    __shared__ unsigned int* glBuffer;
    __shared__ unsigned int base;
    __shared__ unsigned int lock;
    unsigned int warp_id = THID / 32;
    unsigned int lane_id = THID % 32;
    unsigned int i;
    if(THID==0){
        bufTail = 0;
        base = 0;
        lock = 0;
        glBuffer = glBuffers + blockIdx.x*GLBUFFER_SIZE; 
        assert(glBuffer!=NULL);
    }

    unsigned int regTail, regBase;
    
    __syncthreads();

    selectNodesAtLevel(d_p.degrees, V, shBuffer, glBuffer, &bufTail, level);

    syncBlocks(blockCounter);

    // if(level ==  1 && THID == 0)
    //     printf("%d ", bufTail);
    // bufTail is being incrmented within the loop, 
    // warps should process all the nodes added during the execution of loop
    
    
    // for(unsigned int i = warp_id; i<bufTail ; i = warp_id + base){
    // this for loop is a wrong choice, as many threads will exit from the loop checking the condition
    while(true){
        __syncthreads(); //syncthreads must be executed by all the threads
        regBase = base;
        regTail = bufTail;
        __syncthreads();


        if(regBase == regTail) break; // all the threads will evaluate to true at same iteration
        
        i = regBase + warp_id;

        if(THID == 0){
            // update base for next iteration
            base += WARPS_EACH_BLK;
            if(regTail < base )
                base = regTail;
        }
        __syncthreads(); // this call is necessary, so that following update to base is done after everyone get value of i

        
        if(i >= regTail) continue; // this warp won't have to do anything     
        
        
        unsigned int v, start, end;

        v = readFromBuffer(shBuffer, glBuffer, i);
        start = d_p.neighbors_offset[v];
        end = d_p.neighbors_offset[v+1];


        while(true){
            __syncwarp();

            if(start >= end) break;

            unsigned int j = start + lane_id;
            start += WARP_SIZE;
            if(j >= end) continue;

            unsigned int u = d_p.neighbors[j];
            if(ldg(d_p.degrees+u) > level){
                
                unsigned int a = atomicSub(d_p.degrees+u, 1);
            
                if(a == level+1){
                    unsigned int loc = atomicAdd(&bufTail, 1);
                    writeToBuffer(shBuffer, glBuffer, loc, u);
                }

                if(a <= level){
                    // node degree became less than the level after decrementing... 
                    atomicAdd(d_p.degrees+u, 1);
                }
            }
        }

    }

    if(THID == 0 ){
        if(bufTail>0) atomicAdd(global_count, bufTail); // atomic since contention among blocks
        // if(glBuffer!=NULL) free(glBuffer);
    }

}
