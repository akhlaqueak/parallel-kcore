#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>


#include "../inc/gpu_memory_allocation.h"
#include "../inc/device_funcs.h"

int find_kcore(Graph &data_graph,bool write_to_disk){

    G_pointers data_pointers;


    cout<<"Device Copy Started "<<data_graph.V<<data_graph.E<<endl;
    malloc_graph_gpu_memory(data_graph, data_pointers);
    cout<<"Device Copy Done"<<endl;

    unsigned int level = 0;
    unsigned int count = 0;
    unsigned int* global_count  = NULL;
    unsigned int* bufTails  = NULL;
    unsigned int* glBuffers     = NULL;

    chkerr(hipMalloc(&global_count, sizeof(unsigned int)));
    chkerr(hipMalloc(&bufTails, sizeof(unsigned int)*BLK_NUMS));
    hipMemset(global_count, 0, sizeof(unsigned int));
    chkerr(hipMalloc(&glBuffers,sizeof(unsigned int)*BLK_NUMS*GLBUFFER_SIZE));
       
    
	cout<<"K-core Computation Started"<<endl;

    auto start = chrono::steady_clock::now();
    while(count < data_graph.V){
        hipMemset(bufTails, 0, sizeof(unsigned int)*BLK_NUMS);

        selectNodesAtLevel<<<BLK_NUMS, BLK_DIM>>>(data_pointers.degrees, level, 
                        data_graph.V, bufTails, glBuffers);

        processNodes<<<BLK_NUMS, BLK_DIM>>>(data_pointers, level, data_graph.V, 
                        bufTails, glBuffers, global_count);

        chkerr(hipMemcpy(&count, global_count, sizeof(unsigned int), hipMemcpyDeviceToHost));    
        // cout<<"*********Completed level: "<<level<<", global_count: "<<count<<" *********"<<endl;
        level++;
    }
	cout<<"K-core Computation Done"<<endl;

    auto end = chrono::steady_clock::now();
    
    
    // cout << "Elapsed Time: "
    // << chrono::duration_cast<chrono::milliseconds>(end - start).count() << endl;
    // cout <<"MaxK: "<<level-1<<endl;
    
    
	// get_results_from_gpu(data_graph, data_pointers);
    
    hipFree(glBuffers);
    free_graph_gpu_memory(data_pointers);
    // if(write_to_disk){
    //     cout<<"Writing kcore to disk started... "<<endl;
    //     data_graph.writeKCoreToDisk(data_file);
    //     cout<<"Writing kcore to disk completed... "<<endl;
    // }

    return chrono::duration_cast<chrono::milliseconds>(end - start).count();

}


int main(int argc, char *argv[]){
    if (argc < 2) {
        cout<<"Please provide data file"<<endl;
        exit(-1);
    }
    std::string data_file = argv[1];
    bool write_to_disk = false;

    cout<<"Loading Started"<<endl;    
    Graph data_graph(data_file);
    cout<<"Loading Done"<<endl;
    
    vector<int> et;
    for(int i=0;i<REP; i++){
        cout<<"Running iteration: "<<i+1<<endl;
        int t = find_kcore(data_graph, write_to_disk);
        et.push_back(t);
    }
    cout << data_file << " Elapsed Time: ";

    for(auto t: et)
        cout<<t<<" ";
    cout<<(double)accumulate(et.begin(), et.end(), 0)/et.size();
    cout<<endl;
    return 0;
}
