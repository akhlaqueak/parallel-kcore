#include "hip/hip_runtime.h"
/*
 * cuTS:  Scaling Subgraph Isomorphism on Distributed Multi-GPU Systems Using
 *        Trie Based Data Structure
 *
 * Copyright (C) 2021 APPL Laboratories (aravind_sr@outlook.com)
 *
 * This software is available under the MIT license, a copy of which can be
 * found in the file 'LICENSE' in the top-level directory.
 *
 * For further information contact:
 *   (1) Lizhi Xiang (lizhi.xiang@wsu.edu)
 *   (2) Aravind Sukumaran-Rajam (aravind_sr@outlook.com)
 *
 * The citation information is provided in the 'README' in the top-level
 * directory.
 */
#include "../inc/device_funcs.h"
#include "stdio.h"


__device__ void scan(unsigned int *degrees, unsigned int* buffer, unsigned int* e, unsigned int level){
    unsigned int warp_id = threadIdx.x/32;
//    unsigned int lane_id = threadIdx.x%32;
    unsigned int global_threadIdx = blockIdx.x*BLK_DIM + threadIdx.x; 
    printf("a%d--", global_threadIdx);
    for(int i=global_threadIdx; i<d_p.V; i+=N_THREADS){
        if(degrees[i] == level){
            //store this node to shared buffer, at the corresponding warp location
		if(e[warp_id] >= MAX_NE){
            printf("x"); continue;
        }

            unsigned int loc = warp_id*MAX_NE + e[warp_id]; 
            buffer[loc] = i;
            atomicAdd(&e[warp_id], 1); 
		
        }
    }
}

__global__ void PKC(G_pointers &d_p, unsigned int *global_count, int level){


    __shared__ unsigned int buffer[WARPS_EACH_BLK*MAX_NE];
    __shared__ unsigned int e[WARPS_EACH_BLK];



    unsigned int warp_id = threadIdx.x/32;
    unsigned int lane_id = threadIdx.x%32;

  //  unsigned int global_idx = (blockIdx.x)*WARPS_EACH_BLK+warp_id;
//    unsigned int mask = 0xFFFFFFFF;
	printf("c%d", e[warp_id]);

    if(lane_id==0)
        e[warp_id] = 0;

    __syncwarp();

    scan(d_p.degrees, buffer, e, level);
    __syncthreads();

	if(lane_id==0){
	printf("z%d", e[warp_id]);
	}

    for(int i=0; i<e[warp_id]; i++){
        unsigned int v = buffer[warp_id*MAX_NE + i];
        unsigned int start = d_p.neighbors_offset[v];
        unsigned int end = d_p.neighbors_offset[v+1];
        for(int j = start + lane_id; j<end ; j+=32){
            int a = 0;
                printf("%d*", j);
            unsigned int u = d_p.neighbors[j];
            if(d_p.degrees[u] > level){
                a = atomicSub(&d_p.degrees[u], 1);
            }

            if(a == (level+1)){
                int loc = warp_id*MAX_NE + e[warp_id];
                buffer[loc] = u;
                atomicAdd(&e[warp_id], 1);
            }

            if(a <= level){
                atomicAdd(&d_p.degrees[u], 1);
            }
        }

        __syncwarp();
    }

    if(lane_id == 0){
        atomicAdd(&global_count[0], e[warp_id]);    
	}

}
